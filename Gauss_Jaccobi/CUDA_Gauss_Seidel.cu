#include "hip/hip_runtime.h"
// This program is an exemple the vs setup for Cuda
#include <iostream>
#include <chrono>
#include<hip/hip_runtime.h>
#include<>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <fstream>

__global__ void GaussSeidel(float * matrix , float * vector , float * x , int  r,int n, int red_black) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n && i%2 == red_black ) {

		x[i] = vector[i] / (*(matrix + i * n + i));
		
		for (int j = 0; j < n; j++) {
			if (j != i) {
				x[i] -= (*(matrix + i * n + j)*x[j]) / (*(matrix + i * n + i));
			}

		}

	}
}




float * laplacian2d_matrix(int size, int N) {
	float * laplacian;
	laplacian = (float*)malloc(size *size * sizeof(float));
	int side = (int)sqrt((double)size);
	for (int i = 0; i < size*size; i++) {
		laplacian[i] = 0;
	}
	for (int i = 0; i < size; i++) {
		if ((((i + 1) * size + i) < size * size) && (( (i+1) % side) != 0)) {
			laplacian[(i + 1) * size + i] = 1;
		}

		if (0 <= ((i - 1) * size + i) && ((i % side) != 0)) {
			laplacian[(i - 1) * size + i] = 1;
		}

		laplacian[i * size + i] = -4;

		if (((i + N) * size + i) < size * size) {
			laplacian[(i + N) * size + i] = 1;
		}

		if (0 <= ((i - N) * size + i)) {
			laplacian[(i - N) * size + i] = 1;
		}
	}
	return laplacian;
	
}


int main() {

	//laplace solver

	int size = 121; // For a square grid [size * size]

	//host input
	float * laplacian = (float*)malloc(size *size *size *size * sizeof(float));
	float * vector = (float*)malloc(size *size * sizeof(float));
	//device input
	float * laplacian_device;
	float * vector_device;

	//caling generator
	laplacian = laplacian2d_matrix(size*size, size);

	//creating inital values
	for (int i = 0; i < size*size; i++) {
		vector[i] = 0;
	}
	for (int i = 0; i < size; i++) {
		vector[(size - 1)*size + i] = -10;
	}
	//output host
	float * test_laplacian = (float*)malloc(size *size * sizeof(float));

	for (int i = 0; i < size*size; i++) {
		test_laplacian[i] = 0;
	}

	//device output
	float * test_laplacian_device;

	//test_laplacian = jacobi((float*)laplacian, vector, size*size,1e-4F);

	// Allocate memory for each vector on GPU
	hipMalloc(&laplacian_device, size *size *size *size * sizeof(float));
	hipMalloc(&vector_device, size *size * sizeof(float));
	hipMalloc(&test_laplacian_device, size *size * sizeof(float));


	hipMemcpy(laplacian_device, laplacian, size *size *size *size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(vector_device, vector, size *size * sizeof(float), hipMemcpyHostToDevice);
	
	int blockSize, gridSize;

	// Number of threads in each thread block
	blockSize = 64;

	// Number of thread blocks in grid
	gridSize = (int)  (ceil((float)size*size / blockSize));


	// Execute the kernel
	for (int i = 0; i < 1000; i++) {
		GaussSeidel << <gridSize, blockSize >> > (laplacian_device, vector_device, test_laplacian_device, size, size*size, 0);//even index first
		GaussSeidel << <gridSize, blockSize >> > (laplacian_device, vector_device, test_laplacian_device, size, size*size, 1); // odd index
		std::cout << i << std::endl;
		hipDeviceSynchronize(); // sync the result
	}


	//sync
	hipDeviceSynchronize();

	// Copy array back to host
	hipMemcpy(test_laplacian, test_laplacian_device, size *size * sizeof(float), hipMemcpyDeviceToHost);


	//output csv file
	std::ofstream myfile;
	myfile.open("data.csv");
	for (int i = 0; i < size; i++)
	{
		myfile << test_laplacian[i*size + 0];

		for (int j = 1; j < size; j++)
		{
			myfile << ", " << test_laplacian[i*size + j];
		}
		myfile << std::endl;
	}
	myfile.close();

	// Release device memory
	hipFree(vector_device);
	hipFree(laplacian_device);
	hipFree(test_laplacian_device);

	// Release host memory
	free(vector);
	free(laplacian);
	free(test_laplacian);
	return 0;
}